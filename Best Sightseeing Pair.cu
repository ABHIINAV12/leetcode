#include "hip/hip_runtime.h"

#include<bits/stdc++.h>
#define ld long double
#define pb push_back
#define ff first
#define ss second
#define f(i,x,n) for(int i=x;i<(int)n;++i)
#define vi vector<int>
#define vvi vector<vector<int>>
#define vvvi vector<vector<vector<int>>>
#define pq priority_queue<int>
#define pqs priority_queue<int,vi,greater<int>>
#define vpii vector<pair<int,int>>
#define pii pair<int,int>
#define all(x) x.begin(),x.end()
#define sz(x) (int)x.size()
#define mpi map<int,int>
#define ll long long 


class Solution {
public:


    int maxScoreSightseeingPair(vector<int>& a) {
        int n=a.size();
        priority_queue<int> here;
        here.push(a[0]);
        int ret=0;

        f(i,1,n){
            ret=max(ret,a[i]-i + here.top());
            here.push(a[i]+i);
        }

        return ret;
           
    }
};