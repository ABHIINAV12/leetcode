#include "hip/hip_runtime.h"
#include<bits/stdc++.h>
#define ld long double
#define pb push_back
#define ff first
#define ss second
#define f(i,x,n) for(int i=x;i<(int)n;++i)
#define vi vector<int>
#define vvi vector<vector<int>>
#define vvvi vector<vector<vector<int>>>
#define pq priority_queue<int>
#define pqs priority_queue<int,vi,greater<int>>
#define vpii vector<pair<int,int>>
#define pii pair<int,int>
#define all(x) x.begin(),x.end()
#define sz(x) (int)x.size()
#define mpi map<int,int>
#define ll long long 


class Solution {
public:
    int bestTeamScore(vector<int>& s, vector<int>& a) {
        int n=sz(s);
        vpii vt; f(i,0,n) 
            vt.pb({a[i],s[i]});
        sort(all(vt));
        vi dp(n,0); 
        f(i,0,n){
            dp[i]=vt[i].ss;
            for(int j=i-1;j>=0;--j){
                if(vt[j].ss<=vt[i].ss)
                dp[i]=max(dp[i],dp[j]+vt[i].ss);
            }
        }
        return *max_element(all(dp));
    }
};