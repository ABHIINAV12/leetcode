#include "hip/hip_runtime.h"
#include<bits/stdc++.h>
#define ld long double
#define pb push_back
#define ff first
#define ss second
#define f(i,x,n) for(int i=x;i<(int)n;++i)
#define vi vector<int>
#define vvi vector<vector<int>>
#define vvvi vector<vector<vector<int>>>
#define pq priority_queue<int>
#define pqs priority_queue<int,vi,greater<int>>
#define vpii vector<pair<int,int>>
#define pii pair<int,int>
#define all(x) x.begin(),x.end()
#define sz(x) (int)x.size()
#define mpi map<int,int>
#define ll long long 

class Solution {
public:
    int constrainedSubsetSum(vector<int>& a, int k) {
        int n=sz(a);
        vi dp(n);
        stack<pii> h,hh;
        f(i,0,n){
            dp[i]=a[i];
            while(!hh.empty() && (i-hh.top().ff)>k) hh.pop();
            if(hh.empty())
                while(!h.empty())
                    hh.push({h.top().ff, max(hh.empty() ? 0 : hh.top().ss , dp[h.top().ff])}), h.pop();
            dp[i]+=max({0,h.empty() ? 0 : h.top().ss ,hh.empty() ? 0 : hh.top().ss });
            if(h.empty())
                h.push({i,dp[i]});
            else h.push({i,max(dp[i],h.top().ss)});
        }
        return *max_element(all(dp));
    }
};