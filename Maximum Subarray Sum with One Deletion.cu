#include "hip/hip_runtime.h"
#include<bits/stdc++.h>
#define ld long double
#define pb push_back
#define ff first
#define ss second
#define f(i,x,n) for(int i=x;i<(int)n;++i)
#define vi vector<int>
#define vvi vector<vector<int>>
#define vvvi vector<vector<vector<int>>>
#define pq priority_queue<int>
#define pqs priority_queue<int,vi,greater<int>>
#define vpii vector<pair<int,int>>
#define pii pair<int,int>
#define all(x) x.begin(),x.end()
#define sz(x) (int)x.size()
#define mpi map<int,int>
#define ll long long 


class Solution {
public:
    int maximumSum(vector<int>& a) {
        int n=sz(a);
        bool pos=0; f(i,0,n) if(a[i]>=0) pos=1;
        if(!pos)
            return *max_element(all(a));
        int mx=0,curr=0; f(i,0,n){
            curr+=a[i];
            if(curr<0) curr=0;
            mx=max(mx,curr);
        }
        vi left(n,0); curr=0;
        f(i,0,n) {
            curr+=a[i];
            curr=max(curr,0);
            left[i]=curr;
        }
        vi right(n,0); curr=0;
        for(int i=n-1;i>=0;--i){
            curr+=a[i];
            curr=max(curr,0);
            right[i]=curr;
        }
        f(i,1,n-1)
            if(a[i]<0)
                mx=max(mx,left[i-1]+right[i+1]);
            
        return mx;
    }
};